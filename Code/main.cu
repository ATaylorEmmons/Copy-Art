#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>

#include "utils.h"
#include "Rasterizer.h"
#include "CopyArt.h"



#define imin(a,b) (a<b?a:b)
const int N = 1920*1080*3;
const int THREADS = 1024;
const int BLOCKS = 1024;


__global__ void cuda_fitness(uint8_t* imageMemory, uint8_t* targetMemory, uint32_t* storage) {

    __shared__ int cache[THREADS];

    int t_id = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int cacheId = threadIdx.x;


    int store = 0;
    while(t_id < N) {
        store += abs(imageMemory[t_id] - targetMemory[t_id]);
        t_id += stride;
    }

    cache[cacheId] = store;

    __syncthreads();

    int i = blockDim.x/2;

    while( i != 0) {
      if(cacheId < i) {
        cache[cacheId] += cache[cacheId + i];
      }
      __syncthreads();
      i /= 2;
    }

    if(cacheId == 0) {
      storage[blockIdx.x] = cache[0];
    }
}

int main() {

  static const int WIDTH = 1920;
  static const int HEIGHT = 1080;


  RNG rng(Timer::now());

  Image buffer(WIDTH, HEIGHT);
  Image target("hk.png");


  bool useCuda = false;
  int genCount = 100;
  int specCount = 100;
  int rectCount = 5000;

  int minSize = 5;
  int maxSize = 20;

  int elitesToKeep = 4;
  float breedCutOff = .25;

  long startScore = 0;
  long bestScore = 0;

  Timer genTimer;
  long total_time = 0;

  Timer codeTimer;
  long crossover_time = 0;
  long render_time = 0;
  long fitness_time = 0;
  long sort_time = 0;

  std::vector<Specimen> specsA;
  specsA.reserve(specCount);

  std::vector<Specimen> specsB;
  specsB.reserve(specCount);

  std::vector<Specimen>* curSpec;
  std::vector<Specimen>* lastSpec;
  std::vector<Specimen>* tempSpec;

  Color clearColor = Color(0, 0, 0);

  int IMG_MEM_SIZE =  WIDTH*HEIGHT*3*sizeof(uint8_t);
  uint8_t* deviceTarget;
  uint8_t* deviceImage;
  uint32_t* deviceStorage;

  //uint8_t* resultStorage;

  uint32_t* resultStorage;

/*CUDA INIT */

  if(useCuda) {
    //target
    hipMalloc( (void**)&deviceTarget, IMG_MEM_SIZE );
    hipMemcpy( deviceTarget, (uint8_t*)target.getMemoryPtr(), IMG_MEM_SIZE, hipMemcpyHostToDevice);

    resultStorage = (uint32_t *)malloc(IMG_MEM_SIZE);

    //image
    hipMalloc( (void**)&deviceImage, IMG_MEM_SIZE );
    //result
    hipMalloc( (void**)&deviceStorage, BLOCKS*sizeof(uint32_t));
  }


/* END CUDA INIT*/


  genTimer.start();
/* GENERATION 1 */
  for(int i = 0; i < specCount; i++) {

    specsA.push_back(Specimen(rectCount, rng, WIDTH, HEIGHT, minSize, maxSize));
    clear(clearColor, buffer);
    drawRects(specsA[i].traits.data(), rectCount, buffer);
    specsA[i].score = fitness(target, buffer);

    specsB.push_back(specsA[i]);

  }


  curSpec = &specsA;
  lastSpec = &specsB;

  codeTimer.start();
  std::sort(curSpec->begin(), curSpec->end());
  sort_time += codeTimer.stops();

  startScore = curSpec->at(0).score;

  total_time += genTimer.stops();
  println("Initilization: " + std::to_string(total_time/1000) + "ms");

/* THE REST */
  for(int curGen = 0; curGen < genCount; curGen++) {
    genTimer.start();

    for(int i = 0; i < elitesToKeep; i++) {
      curSpec->at(i) = lastSpec->at(i);
    }
    for(int i = elitesToKeep; i < specCount; i++) {

      //Crossover
      codeTimer.start();

      //Only the top half are parents
      int index1 = rng.runifInt(0, specCount*breedCutOff - 1);
      int index2 = rng.runifInt(0, specCount*breedCutOff - 1);

      while(index1 == index2) {
        index2 = rng.runifInt(0, specCount / 2 - 1);
      }

      Specimen* parentA = &lastSpec->at(index1);
      Specimen* parentB = &lastSpec->at(index2);

      curSpec->at(i) =  Specimen(parentA, parentB, rectCount, rng);
      crossover_time += codeTimer.stops();


      //Draw and Score
      codeTimer.start();
      clear(clearColor, buffer);
      drawRects(curSpec->at(i).traits.data(), rectCount, buffer);
      render_time += codeTimer.stops();

      codeTimer.start();

      if(useCuda) {
        //Cuda
        hipMemcpy( deviceImage, (void*)buffer.getMemoryPtr(), IMG_MEM_SIZE, hipMemcpyHostToDevice);
        cuda_fitness <<<BLOCKS, THREADS>>>(deviceImage, deviceTarget, deviceStorage);
        hipMemcpy(resultStorage, deviceStorage, BLOCKS*sizeof(uint32_t), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        uint32_t score = 0;
        for(int i = 0; i < BLOCKS; i++) {
          score += resultStorage[i];
        }

        curSpec->at(i).score = score;

      } else {
        curSpec->at(i).score = fitness(target, buffer);
      }
      fitness_time += codeTimer.stops();
    }



    codeTimer.start();
    std::sort(curSpec->begin(), curSpec->end());
    sort_time += codeTimer.stops();

    long genTime = genTimer.stops();
    total_time += genTime;

    std::string generationString =  " Score: " +  std::to_string(curSpec->at(0).score)
                                  + " Gen Count: " + std::to_string(curGen)
                                  + " Time: " + std::to_string(genTime/1000) +"ms";

    println(generationString);

    tempSpec = curSpec;
    curSpec = lastSpec;
    lastSpec = tempSpec;

  }

  bestScore = curSpec->at(0).score;
  double improvement = 1.0 - (double)bestScore/(double)startScore;
  drawRects(curSpec->at(0).traits.data(), rectCount, buffer);
  buffer.save("Copied.png");

  println("Improvement: " + std::to_string(improvement));
  println("Sort Time: " + std::to_string(sort_time/1000));
  println("Crossover Time: " + std::to_string(crossover_time/1000));
  println("Render Time: " + std::to_string(render_time/1000));
  println("Fitness Time: " + std::to_string(fitness_time/1000));
  println("Total Time: " + std::to_string(total_time/1000));


/* CLEANUP */
  if(useCuda) {
    hipFree(deviceTarget);
    hipFree(deviceImage);
    hipFree(deviceStorage);

    free(resultStorage);
  }
}
















//
